#include "reduce.cuh"
#include <cstdio>

int main(int argc, char *argv[]) {
  int n = atol(argv[1]);
  int threads_per_block = atol(argv[2]);

  auto arr = new int[n];

  for (int i = 0; i < n; i++) {
    arr[i] = 1;
  }

  hipEvent_t start;
  hipEvent_t stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start);
  int result = reduce(arr, n, threads_per_block);
  hipEventRecord(stop);
  hipEventSynchronize(stop);

  if (result != n)
    printf("Error: result is %d instead of %d\n", result, n);

  float ms;
  hipEventElapsedTime(&ms, start, stop);
  printf("%d\n%f\n", result, ms);
}
