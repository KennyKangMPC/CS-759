#include "matmul.cuh"
#include <cstdio>

int main(int argc, char *argv[]) {
  unsigned int n = atol(argv[1]);
  unsigned int block_dim = atol(argv[2]);

  float *A, *B, *C;

  hipMallocManaged((void **)&A, n * n * sizeof(float));
  hipMallocManaged((void **)&B, n * n * sizeof(float));
  hipMallocManaged((void **)&C, n * n * sizeof(float));

  for (size_t i = 0; i < n * n; ++i) {
    A[i] = 2;
    B[i] = 0.5;
  }

  hipEvent_t start;
  hipEvent_t stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start);
  matmul(A, B, C, n, block_dim);
  hipEventRecord(stop);
  hipEventSynchronize(stop);

  for (int i = 0; i < n; i++) {
    for (int j = 0; j < n; j++) {
      if (C[i * n + j] != (float) n){
        printf("Error: (%d, %d) is %f\n", i, j, C[i * n + j]);
      }
    }
  }

  float ms;
  hipEventElapsedTime(&ms, start, stop);
  printf("%f\n%f\n%f\n", C[0], C[n * n - 1], ms);

  hipFree(A);
  hipFree(B);
  hipFree(C);
}
