#include "hip/hip_runtime.h"
#include "mmul.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <random>
#include <hipblas.h>

int main(int argc, char *argv[]) {
	// obtain commandline input
	int n = atol(argv[1]);
  	int n_tests = atol(argv[2]);
  	
  	// set up random number from -1 to 1 generator
	std::random_device entropy_source;
	std::mt19937_64 generator(entropy_source()); 
	const float min = -1.0, max = 1.0; // The range for the random number
	std::uniform_real_distribution<float> dist(min, max);
	
  	// allocate array
	float *a, *b, *c;
	hipMallocManaged((void **)&a, sizeof(float) * n * n);
  	hipMallocManaged((void **)&b, sizeof(float) * n * n);
  	hipMallocManaged((void **)&c, sizeof(float) * n * n);
  	
  	// insert random initial value into it
	for (int i = 0; i < n * n; i++) {
		a[i] = dist(generator);
		b[i] = dist(generator);
		c[i] = dist(generator);
	}
	
	// setup the use of cublas
	// Create a handle for CUBLAS
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	
	/// time for the operations.
	// set up timer
  	hipEvent_t start;
  	hipEvent_t stop;
  	hipEventCreate(&start);
  	hipEventCreate(&stop);
  	
  	hipEventRecord(start);
  	for (int i = 0; i < n_tests; i++) {
  		mmul(handle, a, b, c, n);
  	}
	hipEventRecord(stop);
  	hipEventSynchronize(stop);
  	// Get the elapsed time in milliseconds
	float ms;
	hipEventElapsedTime(&ms, start, stop);
	float averageTime = ms/n_tests;
	printf("%f\n", averageTime);
	
	//clean up everything
	hipblasDestroy(handle);
	hipFree(a);
	hipFree(b);
	hipFree(c);
}
