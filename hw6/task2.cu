#include "hip/hip_runtime.h"
#include "scan.cuh"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <random>

int main(int argc, char *argv[]) {
	int n = atol(argv[1]);
 	int threads_per_block = atol(argv[2]);
 	
 	// set up random number from -1 to 1 generator
	std::random_device entropy_source;
	std::mt19937_64 generator(entropy_source()); 
	const float min = -1.0, max = 1.0; // The range for the random number
	std::uniform_real_distribution<float> dist(min, max);
	
	float *input, *output;
	hipMallocManaged((void **)&input, sizeof(float) * n);
  	hipMallocManaged((void **)&output, sizeof(float) * n);
	
	for (int i = 0; i < n; i++) {
		input[i] = dist(generator);
	}
	
	/// time for the operations.
	// set up timer
  	hipEvent_t start;
  	hipEvent_t stop;
  	hipEventCreate(&start);
  	hipEventCreate(&stop);
  	
  	// start timing
  	hipEventRecord(start);
  	scan(input, output, n, threads_per_block);
  	hipEventRecord(stop);
  	hipEventSynchronize(stop);
  	
  	// Get the elapsed time in milliseconds
	float ms;
	hipEventElapsedTime(&ms, start, stop);
	printf("%f\n%f\n", output[n - 1], ms);
	
	//free memory
	hipFree(input);
	hipFree(output);
}
