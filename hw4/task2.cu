#include "hip/hip_runtime.h"
#include "stencil.cuh"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <random>

int main(int argc, char *argv[]) {
	
	size_t n = atol(argv[1]);
	size_t R = atol(argv[2]);
	size_t threads_per_block = atol(argv[3]);
	
	// set up random number from -1 to 1 generator
	std::random_device entropy_source;
	std::mt19937_64 generator(entropy_source()); 
	const int min = -1.0, max = 1.0; // The range for the random number generator is -1.0 to 1.0
	// there are tons of oter distributino that could be found from https://en.cppreference.com/w/cpp/header/random
	std::uniform_real_distribution<float> dist(min, max);
	
	float *image, *output, *mask;
	size_t mask_size = 2 * R + 1;
	
	// allocate array 
	hipMallocManaged((void **)&image, n * sizeof(float));
  	hipMallocManaged((void **)&output, n * sizeof(float));
  	hipMallocManaged((void **)&mask, mask_size * sizeof(float));
  	
  	for (size_t i = 0; i < n; i++) {
  		image[i] = dist(generator);
  	}
	
	for (size_t i = 0; i < mask_size; ++i) {
    	mask[i] = dist(generator);
  	}
	
	//set up timer
 	hipEvent_t start;
  	hipEvent_t stop;
  	hipEventCreate(&start);
  	hipEventCreate(&stop);
	
	// start timing and test
	hipEventRecord(start);
	stencil(image, mask, output, n, R, threads_per_block);
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float ms;
  	hipEventElapsedTime(&ms, start, stop);
  	printf("%f\n%f\n", output[n - 1], ms);
  	
  	hipFree(image);
  	hipFree(output);
  	hipFree(mask);
}
