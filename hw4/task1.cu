#include "hip/hip_runtime.h"
#include "matmul.cuh"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <random>

int main(int argc, char *argv[]) {
	// obtain user input
	size_t n = atol(argv[1]);
	size_t threads_per_block = atol(argv[2]);
	
	// set up random number from -1 to 1 generator
	std::random_device entropy_source;
	std::mt19937_64 generator(entropy_source()); 
	const float min = -1.0, max = 1.0; // The range for the random number generator is -1.0 to 1.0
	// there are tons of oter distributino that could be found from https://en.cppreference.com/w/cpp/header/random
	std::uniform_real_distribution<float> distA(min, max);
	std::uniform_real_distribution<float> distB(min, max);

	// allocate array
	float *ma, *mb, *mc;
	// device array

	hipMallocManaged((void **)&ma, sizeof(float) * n * n);
  	hipMallocManaged((void **)&mb, sizeof(float) * n * n);
  	hipMallocManaged((void **)&mc, sizeof(float) * n * n);

	// insert random initial value into it
	for (size_t i = 0; i < n * n; i++) {
		ma[i] = distA(generator);
		mb[i] = distB(generator);
	}
	
	// set up timer
  	hipEvent_t start;
  	hipEvent_t stop;
  	hipEventCreate(&start);
  	hipEventCreate(&stop);

  	// record time
  	hipEventRecord(start);
  	matmul(ma, mb, mc, n, threads_per_block);
  	hipEventRecord(stop);
  	hipEventSynchronize(stop);
  	
  	// Get the elapsed time in milliseconds
	float ms;
	hipEventElapsedTime(&ms, start, stop);
	
	// print out the last element of c and the time
  	printf("%f\n%f\n", mc[n * n - 1], ms);
  	
  	// clearn memory
  	hipFree(ma);
  	hipFree(mb);
  	hipFree(mc); 	
}
