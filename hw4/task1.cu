#include "hip/hip_runtime.h"
#include "matmul.cuh"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <random>

int main(int argc, char *argv[]) {
	// obtain user input
	size_t n = atol(argv[1]);
	size_t threads_per_block = atol(argv[2]);
	
	// set up random number from -1 to 1 generator
	std::random_device entropy_source;
	std::mt19937_64 generator(entropy_source()); 
	const int min = -1.0, max = 1.0; // The range for the random number generator is -1.0 to 1.0
	// there are tons of oter distributino that could be found from https://en.cppreference.com/w/cpp/header/random
	std::uniform_real_distribution<float> dist(min, max);
	
	// allocate array
	float *a, *b, *c;
	// device array
	hipMallocManaged((void **)&a, n * n * sizeof(float));
  	hipMallocManaged((void **)&b, n * n * sizeof(float));
  	hipMallocManaged((void **)&c, n * n * sizeof(float));

	// insert random initial value into it
	for (size_t i = 0; i < n * n; i++) {
		a[i] = dist(generator);
		b[i] = dist(generator);
	}
	
	// allocate device
	int device = -1;
	hipGetDevice(&device);
	hipMemPrefetchAsync(a, sizeof(float) * n * n, device, NULL);
  	hipMemPrefetchAsync(b, sizeof(float) * n * n, device, NULL);
	
	// set up timer
  	hipEvent_t start;
  	hipEvent_t stop;
  	hipEventCreate(&start);
  	hipEventCreate(&stop);
  	
  	// record time
  	hipEventRecord(start);
  	matmul(a, b, c, n, threads_per_block);
  	hipEventRecord(stop);
  	hipEventSynchronize(stop);
  	
  	// Get the elapsed time in milliseconds
	float ms;
	hipEventElapsedTime(&ms, start, stop);
	
	// print out the last element of c and the time
  	printf("%f\n%f\n", c[n * n - 1], ms);
  	
  	// clearn memory
  	hipFree(a);
  	hipFree(b);
  	hipFree(c); 	
}
